#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_
#include <stdio.h>
// P. Bauer, 2011

// Must be identifical with GridDim
#define SHARED_DIM_X  22
#define SHARED_DIM_Y  22

__shared__ unsigned int as[(SHARED_DIM_X)*(SHARED_DIM_Y)];
__shared__ unsigned int bs[(SHARED_DIM_X)*(SHARED_DIM_Y)];
__shared__ unsigned int cs[(SHARED_DIM_X)*(SHARED_DIM_Y)];
__shared__ unsigned int ds[(SHARED_DIM_X)*(SHARED_DIM_Y)];
__shared__ unsigned int es[(SHARED_DIM_X)*(SHARED_DIM_Y)];
__shared__ unsigned int fs[(SHARED_DIM_X)*(SHARED_DIM_Y)];

__device__ unsigned int iLin( int r, int c, int m, int n) {
    r = ( r >= m ? r - m : r );
    r = ( r <  0 ? r + m : r );
 
    c = ( c >= n ? c - n : c );
    c = ( c <  0 ? c + n : c );
    return c * m + r;
}

__global__ void runLGCA(unsigned int* b1, unsigned int* b2, unsigned int* b3, unsigned int* b4, unsigned int* b5, unsigned int* b6, int m, int n)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int tidx=threadIdx.x+1;
	int tidy=threadIdx.y+1;

	// Adressing in shared memory
	int linidx = iLin(tidx,tidy, SHARED_DIM_X, SHARED_DIM_Y);
	// Adressing in global memory
	int linidxG = iLin(x,y,m,n);
	
	// Actual cell
	as[linidx] = b1[linidxG];
	bs[linidx] = b2[linidxG];
	cs[linidx] = b3[linidxG];
	ds[linidx] = b4[linidxG];
	es[linidx] = b5[linidxG];
	fs[linidx] = b6[linidxG];

	// Ghost cells
	if ( threadIdx.x == 0 && threadIdx.y != blockDim.y - 1 ) // Upper left corner
	{
		fs[iLin(0, tidy,      SHARED_DIM_X, SHARED_DIM_Y)] = b6[iLin(x-1, y,   m, n)];
		es[iLin(0, tidy+1, SHARED_DIM_X, SHARED_DIM_Y)] = b5[iLin(x-1, y+1, m, n)];
	}
	if ( threadIdx.x == blockDim.x - 1 && threadIdx.y != 0 ) // Right lower corner
	{
		bs[iLin(blockDim.x + 1, tidy-1,   SHARED_DIM_X, SHARED_DIM_Y)] = b2[iLin(x+1, y-1, m, n)];
		cs[iLin(blockDim.x + 1, tidy,      SHARED_DIM_X, SHARED_DIM_Y)] = b3[iLin(x+1, y,   m, n)];
	}
	if ( threadIdx.y == 0 && threadIdx.x != blockDim.x - 1) // Left upper corner
	{
		as[iLin(tidx,    0, SHARED_DIM_X, SHARED_DIM_Y)] = b1[iLin(x, y-1, m,n)];
		bs[iLin(tidx+1, 0, SHARED_DIM_X, SHARED_DIM_Y)] = b2[iLin(x+1, y-1, m, n)];
	}
	if ( threadIdx.y == blockDim.y - 1  && threadIdx.x != 0) // lower corner
	{
		ds[iLin(tidx,    blockDim.y + 1, SHARED_DIM_X, SHARED_DIM_Y)] = b4[iLin(x,  y+1, m, n)];
		es[iLin(tidx-1,   blockDim.y + 1, SHARED_DIM_X, SHARED_DIM_Y)] = b5[iLin(x-1,y+1, m, n)];
	}
	if ( threadIdx.x == 0 && threadIdx.y == blockDim.y - 1) // Left lower corner
	{
		fs[iLin(0,  blockDim.y    ,   SHARED_DIM_X, SHARED_DIM_Y)] = b6[iLin(x-1, y,   m, n)];
		es[iLin(0, blockDim.y + 1, SHARED_DIM_X, SHARED_DIM_Y)] = b5[iLin(x-1 ,y+1, m, n)];
		ds[iLin(1, blockDim.y + 1, SHARED_DIM_X, SHARED_DIM_Y)] = b4[iLin(x ,  y+1, m, n)];
	}
	if ( threadIdx.x == blockDim.x - 1 &&  threadIdx.y == 0) // Right upper corner
	{
		as[iLin(blockDim.x   , 0, SHARED_DIM_X, SHARED_DIM_Y)] = b1[iLin(x,   y-1, m, n)];
		bs[iLin(blockDim.x +1, 0, SHARED_DIM_X, SHARED_DIM_Y)] = b2[iLin(x+1, y-1, m, n)];
		cs[iLin(blockDim.x +1, 1, SHARED_DIM_X, SHARED_DIM_Y)] = b3[iLin(x+1, y,   m, n)];
	}
	__syncthreads();

	// Wait for all collisions to be computed and prepare for writing propagation

	unsigned int nextA = as[ iLin(tidx,     tidy-1,   SHARED_DIM_X, SHARED_DIM_Y) ];
	unsigned int nextB = bs[ iLin(tidx+1, tidy-1,   SHARED_DIM_X, SHARED_DIM_Y) ];
	unsigned int nextC = cs[ iLin(tidx+1, tidy,      SHARED_DIM_X, SHARED_DIM_Y) ];
	unsigned int nextD = ds[ iLin(tidx,     tidy+1,  SHARED_DIM_X, SHARED_DIM_Y) ];
	unsigned int nextE = es[ iLin(tidx-1,  tidy+1,  SHARED_DIM_X, SHARED_DIM_Y) ];
	unsigned int nextF = fs[ iLin(tidx-1,   tidy,      SHARED_DIM_X, SHARED_DIM_Y) ];
	
	// Collission protection
	__syncthreads(); 

	as[linidx] = nextA;
	bs[linidx] = nextB;
	cs[linidx] = nextC;
	ds[linidx] = nextD;
	es[linidx] = nextE;
	fs[linidx] = nextF;
	
	__syncthreads();
	
	if ( x < n && y < m ) 
	{
		b1[linidxG] = as[linidx];
		b2[linidxG] = bs[linidx];
		b3[linidxG] = cs[linidx];
		b4[linidxG] = ds[linidx];
		b5[linidxG] = es[linidx];
		b6[linidxG] = fs[linidx];
	}
}


#endif // #ifndef _MATRIXMUL_KERNEL_H_
